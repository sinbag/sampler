#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <utils_cuda.h>
#include <point2d.h>
#include <vector>
#include <thrust/complex.h>

__global__
void fouriermd_compute_kernel(double * d_outReal, double * d_outImag,
                              double* d_inPoints, double* d_inFuncVals,
                              int npts, int resolution, int ndims,
                              double twopi, float frequencyStep){
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;

    //printf("thread indices %d , %d \n", m , l);

    if ( col >= resolution ||  row >= resolution)// || s >= points.size())
    {
        return;
    }
    else{
        double realCoeff = 0.0, imagCoeff = 0.0;
        __syncthreads();
        int halfRes = resolution * 0.5;

        if(ndims == 2){
            int wy = (row - halfRes) * frequencyStep;
            int wx = (col - halfRes) * frequencyStep;

            for(int i = 0; i < npts; i++){
                double exp = -twopi * (wx * d_inPoints[2*i] + wy * d_inPoints[2*i+1]);
                realCoeff += d_inFuncVals[i] * cosf(exp);
                imagCoeff += d_inFuncVals[i] * sinf(exp);
            }
            __syncthreads();
            int index = row * resolution + col;
            //printf("I am thread %d , %d in block %d , %d \n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
            d_outReal[index] = realCoeff;
            d_outImag[index] = imagCoeff;
        }
        else if(ndims == 3){
            int s = threadIdx.z + blockDim.z * blockIdx.z;
            int wy = (row - halfRes) * frequencyStep;
            int wx = (col - halfRes) * frequencyStep;
            int wz = (s - halfRes) * frequencyStep;

            for(int i = 0; i < npts; i++){
                double exp = -twopi * (wx * d_inPoints[3*i] + wy * d_inPoints[3*i+1] + wz * d_inPoints[3*i+2]);
                realCoeff += d_inFuncVals[i] * cosf(exp);
                imagCoeff += d_inFuncVals[i] * sinf(exp);
            }
            __syncthreads();
            int index = s*resolution*resolution + row * resolution + col;
            //printf("I am thread %d , %d in block %d , %d \n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
            d_outReal[index] = realCoeff;
            d_outImag[index] = imagCoeff;
        }
        else if(ndims == 4){
            int s = threadIdx.z + blockDim.z * blockIdx.z;
            int wy = (row - halfRes) * frequencyStep;
            int wx = (col - halfRes) * frequencyStep;
            int wz = (s - halfRes) * frequencyStep;

            __syncthreads();
            for(int u = 0; u < resolution; u++)
            //int u = halfRes;
            {
                int wu = (u - halfRes) * frequencyStep;
                //printf("\n %d, %d, %d, %d %d: ", wx, wy, wz, wu, s);
                for(int i = 0; i < npts; i++){
                    double exp = -twopi * (wx * d_inPoints[4*i] +
                                           wy * d_inPoints[4*i+1] +
                                           wz * d_inPoints[4*i+2] +
                                           wu * d_inPoints[4*i+3]);

                    realCoeff += d_inFuncVals[i] * cosf(exp);
                    imagCoeff += d_inFuncVals[i] * sinf(exp);
                }
                __syncthreads();
                int index = u*resolution*resolution*resolution +
                            s*resolution*resolution +
                            row * resolution + col;
                //printf("I am thread %d , %d in block %d , %d \n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
                d_outReal[index] = realCoeff;
                d_outImag[index] = imagCoeff;
                __syncthreads();
            }
        }
        else if(ndims == 5){
            int s = threadIdx.z + blockDim.z * blockIdx.z;
            int wy = (row - halfRes) * frequencyStep;
            int wx = (col - halfRes) * frequencyStep;
            int wz = (s - halfRes) * frequencyStep;

            for(int u = 0; u< resolution; u++){
                for(int v = 0; v < resolution; v++){
                    int wu = (u - halfRes) * frequencyStep;
                    int wv = (v - halfRes) * frequencyStep;
                    for(int i = 0; i < npts; i++){
                        double exp = -twopi * (wx * d_inPoints[5*i] +
                                               wy * d_inPoints[5*i+1] +
                                               wz * d_inPoints[5*i+2] +
                                               wu * d_inPoints[5*i+3] +
                                               wv * d_inPoints[5*i+4]);

                        realCoeff += d_inFuncVals[i] * cosf(exp);
                        imagCoeff += d_inFuncVals[i] * sinf(exp);
                    }
                    __syncthreads();
                    int index = v*resolution*resolution*resolution*resolution +
                                u*resolution*resolution*resolution +
                                s*resolution*resolution +
                                row * resolution + col;
                    //printf("I am thread %d , %d in block %d , %d \n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
                    d_outReal[index] = realCoeff;
                    d_outImag[index] = imagCoeff;
                }
            }
        }
      ///Do not uncomment!!!
      ///Division by N for real and Imaginaru coeffs is done in the
      /// C++ Host machine FourierAnalyzer code
      //realCoeffs /= N; imagCoeffs /= N;

    __syncthreads();
    // if(row == 384 && col == 384)
    //     printf("\nreal %f and imag %f : resolution: %d %d", realCoeff, imagCoeff, numRows, numCols);
    // __syncthreads();
  }
}

void cftmd_cuda(double * d_outReal, double * d_outImag,
              double* d_inPoints,  double* d_inFuncVals,
              int npts, int resolution, int ndims,
              double twopi, float frequencyStep){
    hipDeviceSynchronize();
    //printf("Image Size: %d %d \n", numRows, numCols);

    if(ndims == 2)
    {
    const dim3 blockThreadSize(16, 16);

    int bx = (resolution + blockThreadSize.x - 1 ) / blockThreadSize.x;
    int by = (resolution + blockThreadSize.y - 1 ) / blockThreadSize.y;
    dim3 gridBlockSize( bx, by, 1);

    fouriermd_compute_kernel<<<gridBlockSize, blockThreadSize>>>(d_outReal, d_outImag, d_inPoints,
      d_inFuncVals, npts, resolution, ndims, twopi, frequencyStep);
    }
    else{
      const dim3 blockThreadSize(8,8,8);

      int bx = (resolution + blockThreadSize.x - 1 ) / blockThreadSize.x;
      int by = (resolution + blockThreadSize.y - 1 ) / blockThreadSize.y;
      int bz = (resolution + blockThreadSize.z - 1 ) / blockThreadSize.z;
      dim3 gridBlockSize( bx, by, bz);

      fouriermd_compute_kernel<<<gridBlockSize, blockThreadSize>>>(d_outReal, d_outImag, d_inPoints,
        d_inFuncVals, npts, resolution, ndims, twopi, frequencyStep);
  }
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <utils_cuda.h>
#include <point2d.h>
#include <vector>
#include <thrust/complex.h>
//#include <VarianceFormulationAnalyzer.h>


// __global__ void atomic_continuous_fourier_transform(float *d_outReal, float *d_outImag,
//                                             std::vector<Point2d> &points, double twopi){
//     int myId = threadIdx.x + blockDim.x * blockIdx.x;
//     int tid = threadIdx.x;
//
//     if(m >= points.size()){
//       return;
//     }
//     else{
//       double exp = -twopi * (wx * points[myId].x + wy * points[myId].y);
//       atomicAdd(d_outReal, cos(exp));
//       atomicAdd(d_outImag, sin(exp));
//     }
// }

__global__
void fourier_compute_kernel(double * d_outReal, double * d_outImag,
                              double* d_inPoints, double* d_inFuncVals,
                              int npts, int numCols,
                              int numRows, double twopi, float frequencyStep){
  int row = threadIdx.x + blockDim.x * blockIdx.x;
  int col = threadIdx.y + blockDim.y * blockIdx.y;
  //int s = threadIdx.z + blockDim.z * blcokIdx.z;
  //printf("thread indices %d , %d \n", m , l);

  if ( col >= numCols ||  row >= numRows)// || s >= points.size())
    {
        return;
    }
 else{
      double realCoeff = 0.0, imagCoeff = 0.0;
      __syncthreads();
      int half_xRes = numCols * 0.5;
      int half_yRes = numRows * 0.5;
      int wy = (row - half_yRes) * frequencyStep;
      int wx = (col - half_xRes) * frequencyStep;
      for(int i = 0; i < npts; i++){
          double exp = -twopi * (wx * d_inPoints[2*i] + wy * d_inPoints[2*i+1]);
          realCoeff += d_inFuncVals[i] * cosf(exp);
          imagCoeff += d_inFuncVals[i] * sinf(exp);
      }
      ///Do not uncomment!!!
      ///Division by N for real and Imaginaru coeffs is done in the
      /// C++ Host machine FourierAnalyzer code
      //realCoeffs /= N; imagCoeffs /= N;

    __syncthreads();
    int index = row * numCols + col;
    //printf("I am thread %d , %d in block %d , %d \n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
    d_outReal[index] = realCoeff;
    d_outImag[index] = imagCoeff;
    __syncthreads();
    // if(row == 384 && col == 384)
    //     printf("\nreal %f and imag %f : resolution: %d %d", realCoeff, imagCoeff, numRows, numCols);
    // __syncthreads();
  }
}

void continuous_fourier_transform_cuda(double * d_outReal, double * d_outImag,
                              double* d_inPoints,  double* d_inFuncVals,
                              int npts, int numCols,
                              int numRows, double twopi, float frequencyStep){
    hipDeviceSynchronize();
    //printf("Image Size: %d %d \n", numRows, numCols);

    const dim3 blockSize(16, 16);

    int bx = (numCols + blockSize.x - 1 ) / blockSize.x;
    int by = (numRows + blockSize.y - 1 ) / blockSize.y;
    dim3 gridSize( bx, by, 1);

    fourier_compute_kernel<<<gridSize, blockSize>>>(d_outReal, d_outImag, d_inPoints,
      d_inFuncVals, npts, numCols, numRows, twopi, frequencyStep);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
